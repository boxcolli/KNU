//#include "./common.cpp"

#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel program for the device (GPU): compiled by NVCC
__global__ void add_kernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// main program for the CPU: compiled by MS-VC++
int main()
{
    // host-side data
    const int SIZE = 5;
    const int a[SIZE] = {1, 2, 3, 4, 5};
    const int b[SIZE] = {10, 20, 30, 40, 50};
    int c[SIZE] = {0};

    // device-side data
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    
    // allocate device memory
    hipMalloc((void**)&dev_a, SIZE*sizeof(int));
    hipMalloc((void**)&dev_b, SIZE*sizeof(int));
    hipMalloc((void**)&dev_c, SIZE*sizeof(int));

    // copy from host to device
    hipMemcpy(dev_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);

    // launch a kernel on the GPU with one thread for each element.
    add_kernel<<<1, SIZE>>>(dev_c, dev_a, dev_b);
    hipDeviceSynchronize();

    // copy from device to host
    hipMemcpy(c, dev_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // print
    printf("%d %d %d %d %d + %d %d %d %d %d = %d %d %d %d %d\n",
            a[0], a[1], a[2], a[3], a[4],
            b[0], b[1], b[2], b[3], b[4],
            c[0], c[1], c[2], c[3], c[4]);
    
    // done
    fflush(stdout);
    return 0;
}