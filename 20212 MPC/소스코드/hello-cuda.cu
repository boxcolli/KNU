#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#include <stdio.h>

__global__ void hello()
{
    printf("hello CUDA #%d!\n", threadIdx.x);
}

int main()
{
    hello<<<1,8>>>();
#if defined(__linux__)
    hipDeviceSynchronize();
#endif
    return 0;
}