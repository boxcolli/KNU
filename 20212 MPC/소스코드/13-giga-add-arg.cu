#include "hip/hip_runtime.h"
#include "./common.cpp"

chrono::system_clock::time_point __time_begin[8] = { chrono::system_clock::now(), };

unsigned vecSize = 256 * 1024 * 1024;

__global__ void kernelVecAdd(float *c, const float* a, const float* b, unsigned n, long long* times)
{
	clock_t start = clock();
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		c[i] = a[i] + b[i];
	clock_t end = clock();
	if (i == 0)
		times[0] = (long long)(end - start);
}

int main(const int argc, const char* argv[])
{
	// argv processing
	char* pEnd = nullptr;
	switch (argc)
	{
	case 1:
		break;
	case 2:
		vecSize = strtol(argv[1], &pEnd, 10);
		break;
	default:
		printf("usage: %s [size]\n", argv[0]);
		exit(EXIT_FAILURE);
		break;
	}

	float* vecA = new float[vecSize];
	float* vecB = new float[vecSize];
	float* vecC = new float[vecSize];
	long long* host_times = new long long[1];

	srand(0);
	setNormalizedRandomData(vecA, vecSize);
	setNormalizedRandomData(vecB, vecSize);

	float* dev_vecA;
	float* dev_vecB;
	float* dev_vecC;
	long long* dev_times = nullptr;

	hipMalloc((void**)&dev_vecA, vecSize * sizeof(float));
	hipMalloc((void**)&dev_vecB, vecSize * sizeof(float));
	hipMalloc((void**)&dev_vecC, vecSize * sizeof(float));
	hipMalloc((void**)&dev_times, 1 * sizeof(long long));

	ELAPSED_TIME_BEGIN(1);

	hipMemcpy(dev_vecA, vecA, vecSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vecB, vecB, vecSize * sizeof(float), hipMemcpyHostToDevice);

	ELAPSED_TIME_BEGIN(0);
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid((vecSize + dimBlock.x - 1) / dimBlock.x, 1, 1);
	kernelVecAdd <<<dimGrid, dimBlock >>> (dev_vecC, dev_vecA, dev_vecB, SIZE, dev_times);
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);

	hipMemcpy(vecC, dev_vecC, vecSize * sizeof(float), hipMemcpyDeviceToHost);
	
	ELAPSED_TIME_END(1);

	hipMemcpy(host_times, dev_times, 1 * sizeof(long long), hipMemcpyDeviceToHost);

	// kernel clock calculation
	int peak_clk = 1;
	hipDeviceGetAttribute(&peak_clk, hipDeviceAttributeClockRate, 0);
	printf("num clock = %lld, peak clock rate = %dkHz, elapsed time: %f usec\n",
		host_times[0], peak_clk, host_times[0] * 1000.0f / (float)peak_clk);

	float sumA = getSum(vecA, vecSize);
	float sumB = getSum(vecB, vecSize);
	float sumC = getSum(vecC, vecSize);
	float diff = fabsf(sumC - (sumA + sumB));
	printf("SIZE = %d\n", vecSize);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("sumC = %f\n", sumC);
	printf("diff(sumC, sumA + sumB) = %f\n", diff);
	printf("diff(sumC, sumA + sumB) / SIZE = %f\n", diff / vecSize);
	printVec("vecA", vecA, vecSize);
	printVec("vecB", vecB, vecSize);
	printVec("vecC", vecC, vecSize);

	return 0;
}