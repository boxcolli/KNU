#include "hip/hip_runtime.h"
#include "./common.cpp"

chrono::system_clock::time_point __time_begin[8] = { chrono::system_clock::now(), };

const unsigned SIZE = 256 * 1024 * 1024;

__global__ void singleKernelVecAdd(float* c, const float* a, const float* b)
{
	for (register unsigned i = 0; i < SIZE; ++i)
		c[i] = a[i] + b[i];
}

int main()
{
	float* vecA = new float[SIZE];
	float* vecB = new float[SIZE];
	float* vecC = new float[SIZE];

	srand(0);
	setNormalizedRandomData(vecA, SIZE);
	setNormalizedRandomData(vecB, SIZE);

	float* dev_vecA;
	float* dev_vecB;
	float* dev_vecC;

	hipMalloc((void**)&dev_vecA, SIZE * sizeof(float));
	hipMalloc((void**)&dev_vecB, SIZE * sizeof(float));
	hipMalloc((void**)&dev_vecC, SIZE * sizeof(float));

	ELAPSED_TIME_BEGIN(1);

	hipMemcpy(dev_vecA, vecA, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vecB, vecB, SIZE * sizeof(float), hipMemcpyHostToDevice);

	ELAPSED_TIME_BEGIN(0);
	singleKernelVecAdd <<<1, 1 >>> (dev_vecC, dev_vecA, dev_vecB);
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);

	hipMemcpy(vecC, dev_vecC, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	
	ELAPSED_TIME_END(1);

	float sumA = getSum(vecA, SIZE);
	float sumB = getSum(vecB, SIZE);
	float sumC = getSum(vecC, SIZE);
	float diff = fabsf(sumC - (sumA + sumB));
	printf("SIZE = %d\n", SIZE);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("sumC = %f\n", sumC);
	printf("diff(sumC, sumA + sumB) = %f\n", diff);
	printf("diff(sumC, sumA + sumB) / SIZE = %f\n", diff / SIZE);
	printVec("vecA", vecA, SIZE);
	printVec("vecB", vecB, SIZE);
	printVec("vecC", vecC, SIZE);

	return 0;
}