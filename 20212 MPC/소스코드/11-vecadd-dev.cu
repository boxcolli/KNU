#include "hip/hip_runtime.h"
#include "./common.cpp"

const unsigned SIZE = 1024 * 1024;

__global__ void kernelVecAdd(float* c, const float* a, const float* b, unsigned n)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}55

int main()
{
    float* vecA = new float[SIZE];
    float* vecB = new float[SIZE];
    float* vecC = new float[SIZE];

    setNormalizedRandomData(vecA, SIZE);
    setNormalizedRandomData(vecB, SIZE);

    float* dev_vecA = nullptr;
    float* dev_vecB = nullptr;
    float* dev_vecC = nullptr;

    hipMalloc((void**)&dev_vecA, SIZE*sizeof(float));
    hipMalloc((void**)&dev_vecB, SIZE*sizeof(float));
    hipMalloc((void**)&dev_vecC, SIZE*sizeof(float));

    ELAPSED_TIME_BEGIN(1);

    hipMemcpy(dev_vecA, vecA, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_vecB, vecB, SIZE*sizeof(float), hipMemcpyHostToDevice);

    ELAPSED_TIME_BEGIN(0);

    kernelVecAdd<<<SIZE/1024,1024>>>(dev_vecC, dev_vecA, dev_vecB, SIZE);
    hipDeviceSynchronize();

    ELAPSED_TIME_END(0);
 
    hipMemcpy(vecC, dev_vecC, SIZE*sizeof(float), hipMemcpyDeviceToHost);
    ELAPSED_TIME_END(1);

    float sumA = getSum(vecA, SIZE);
    float sumB = getSum(vecB, SIZE);
    float sumC = getSum(vecC, SIZE);
    float diff = fabsf(sumC - (sumA + sumB));
    printf("SIZE = %d\n", SIZE);
    printf("sumA = %f\n", sumA);
    printf("sumB = %f\n", sumB);
    printf("sumC = %f\n", sumC);
    printf("diff(sumC, sumA + sumB) = %f\n", diff);
    printf("diff(sumC, sumA + sumB) / SIZE = %f\n", diff / SIZE);
    printVec("vecA", vecA, SIZE);
    printVec("vecB", vecB, SIZE);
    printVec("vecC", vecC, SIZE);

    delete[] vecA;
    delete[] vecB;
    delete[] vecC;

    return 0;
}