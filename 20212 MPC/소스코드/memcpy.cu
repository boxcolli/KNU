#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


int main()
{
    // host-side data
    const int SIZE=8;
    const float a[SIZE]={1.,2.,3.,4.,5.,6.,7.,8};
    float b[SIZE]={0.,0.,0.,0.,0.,0.,0.,0};

    // print source
    printf("a={%f,%f,%f,%f,%f,%f,%f,%f}\n", a[0],a[1],a[2],a[3],a[4],a[5],a[6],a[7]);
    fflush(stdout);

    // device-size data
    float* dev_a=nullptr;
    float* dev_b=nullptr;

    // allocate device memory
    hipMalloc((void**)&dev_a, SIZE*sizeof(float));
    hipMalloc((void**)&dev_b, SIZE*sizeof(float));

    // copy
    hipMemcpy(dev_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, dev_a, SIZE*sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(b, dev_b, SIZE*sizeof(float), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);

    // print result
    printf("b={%f,%f,%f,%f,%f,%f,%f,%f}\n", b[0],b[1],b[2],b[3],b[4],b[5],b[6],b[7]);
    fflush(stdout);

    return 0;
}