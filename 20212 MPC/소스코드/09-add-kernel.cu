
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK_ERROR() \
    hipError_t e = hipGetLastError(); \
    if (hipSuccess != e) \
    { \
        printf("cuda failure \"%s\" at %s:%d\n", \
            hipGetErrorString(e), __FILE__, __LINE__); \
        exit(1); \
    }

#define CUDA_CHECK_ERROR()

// CUDA kernel function
__global__ void add_kernel(float* b, const float* a)
{
    int i = threadIdx.x;
    b[i] = a[i] + 1.0f;
}

int main()
{
    // host-side data
    const int SIZE = 8;
    const float a[SIZE] = {0., 1., 2., 3., 4., 5., 6., 7.};
    float b[SIZE] = {0., 0., 0., 0., 0., 0., 0., 0.};

    // print source
    printf("a = {%f, %f, %f, %f, %f, %f, %f, %f}\n",
            a[0], a[1], a[2], a[3], a[4], a[5], a[6], a[7]);
    
    // device-side data
    float* dev_a = nullptr;
    float* dev_b = nullptr;
    
    // allocate device memory
    hipMalloc((void**)&dev_a, SIZE*sizeof(float));
    hipMalloc((void**)&dev_b, SIZE*sizeof(float));
    hipMemcpy(dev_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);

    // kernel
    add_kernel<<<1, SIZE>>>(dev_b, dev_a);
    hipDeviceSynchronize();

    // print
    hipMemcpy(b, dev_b, SIZE*sizeof(float), hipMemcpyDeviceToHost);
    printf("b = {%f, %f, %f, %f, %f, %f, %f, %f}\n",
            b[0], b[1], b[2], b[3], b[4], b[5], b[6], b[7]);
    
    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);

    // error check
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        printf("CUDA: ERROR: cuda failure \"%s\"\n", hipGetErrorString(err));
        exit(1);
    }
    else
    {
        printf("CUDA: success\n");
    }

    // done
    return 0;
}