#include "hip/hip_runtime.h"
#include "./common.cpp"

chrono::system_clock::time_point __time_begin[8] = { chrono::system_clock::now(), };

const unsigned num = 16 * 1024 * 1024;
unsigned blocksize = 1024;

__global__ void kernelAdjDiff(float* b, const float* a, unsigned num)
{
	extern __shared__ float s_data[]; // tile
	register unsigned tx = threadIdx.x; // local index
	register unsigned i = blockIdx.x * blockDim.x + threadIdx.x; // global  index
	if (i < num)
	{
		s_data[tx] = a[i];
		__syncthreads();
		if (tx > 0)
			b[i] = s_data[tx] - s_data[tx - 1];
		else if (i > 0) // the first element of tile
			b[i] = s_data[tx] - a[i - 1];
		else // the first element of entire local memory
			b[i] = s_data[tx] - 0.0f;
	}
}

__device__ float dev_vecA[num];
__device__ float dev_vecB[num];

int main()
{
	float* vecA = new float[num];
	float* vecB = new float[num];

	srand(0);
	setNormalizedRandomData(vecA, num);
	setNormalizedRandomData(vecB, num);

	int deviceCount = 0;
	hipDeviceProp_t deviceProp;
	int driverVersion = 0;
	int runtimeVersion = 0;
	hipGetDeviceCount(&deviceCount);
	hipGetDeviceProperties(&deviceProp, 0);
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);

	ELAPSED_TIME_BEGIN(1);

	void* p_dev_vecA = nullptr;
	void* p_dev_vecB = nullptr;

	hipGetSymbolAddress(&p_dev_vecA, HIP_SYMBOL(dev_vecA));
	hipGetSymbolAddress(&p_dev_vecB, HIP_SYMBOL(dev_vecB));
	hipMemcpy(p_dev_vecA, vecA, num * sizeof(float), hipMemcpyHostToDevice);

	ELAPSED_TIME_BEGIN(0);
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid((num + dimBlock.x - 1) / dimBlock.x, 1, 1);
	blocksize = deviceProp.maxThreadsPerBlock;
	kernelAdjDiff << <dimGrid, dimBlock, blocksize * sizeof(float) >> > ((float*)p_dev_vecB, (float*)p_dev_vecA, num);
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);

	hipMemcpy(vecB, p_dev_vecB, num * sizeof(float), hipMemcpyDeviceToHost);

	ELAPSED_TIME_END(1);

	float sumA = getSum(vecA, num);
	float sumB = getSum(vecB, num);
	printf("SIZE = %d\n", num);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printVec("vecA", vecA, num);
	printVec("vecB", vecB, num);

	return 0;
}